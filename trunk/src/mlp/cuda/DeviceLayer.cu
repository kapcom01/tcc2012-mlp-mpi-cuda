#include "hip/hip_runtime.h"
#include "mlp/cuda/DeviceLayer.h"

#define CUDA_RAND_MAX 4294967295

namespace ParallelMLP
{

__device__
float d_random(hiprandState* state);

__device__
float d_activate(float x);

__device__
float d_derivate(float y);

//===========================================================================//

DeviceLayer::DeviceLayer(uint inUnits, uint outUnits)
{
	init(inUnits, outUnits);
}

//===========================================================================//

__global__
void initRandState(hiprandState* state, int seed, uint connUnits)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < connUnits)
		hiprand_init(seed + i, 0, 0, &state[i]);
}

//===========================================================================//

void DeviceLayer::init(uint inUnits, uint outUnits)
{
	this->inUnits = inUnits + 1;
	this->outUnits = outUnits;
	this->connUnits = (inUnits + 1) * outUnits;
	this->connBlocks = connUnits / TPB + 1;
	this->outBlocks = outUnits / TPB + 1;

	hipMalloc(&weights, connUnits * sizeof(float));
	hipMalloc(&gradient, outUnits * sizeof(float));
	hipMalloc(&funcSignal, (outUnits + 1) * sizeof(float));
	hipMalloc(&errorSignal, inUnits * sizeof(float));
	hipMalloc(&state, connUnits * sizeof(hiprandState));

	float aux = 1;

	hipMemcpy(&funcSignal[outUnits], &aux, sizeof(float),
			hipMemcpyHostToDevice);

	initRandState<<<connBlocks, TPB>>>(state, rand(), connUnits);
}

//===========================================================================//

DeviceLayer::~DeviceLayer()
{

}

//===========================================================================//

__global__
void randomizeWeight(float* weights, hiprandState* state, uint connUnits)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < connUnits)
		weights[i] = d_random(&state[i]);
}

//===========================================================================//

void DeviceLayer::randomize()
{
	randomizeWeight<<<connBlocks, TPB>>>(weights, state, connUnits);
}

//===========================================================================//

__global__
void feedforwardSum(const float* input, float* weights, uint inUnits,
		uint connUnits, float* funcSignal)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = i % inUnits;
	int k = i / inUnits;

	if (i < connUnits)
		funcSignal[k] += weights[i] * input[j];
		//atomicAdd(&funcSignal[k], weights[i] * input[j]);
}

//===========================================================================//

__global__
void feedforwardActivate(float* weights, uint outUnits, float* funcSignal)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < outUnits)
		funcSignal[i] = d_activate(funcSignal[i]);
}

//===========================================================================//

void DeviceLayer::feedforward(const float* input)
{
	this->input = input;

	// Inicializa o sinal funcional
	hipMemset(funcSignal, 0, outUnits * sizeof(float));

	// Calcula as somas ponderadas das entradas
	feedforwardSum<<<connBlocks, TPB>>>(input, weights, inUnits, connUnits,
			funcSignal);

	// Ativa as saídas de cada neurônio
	feedforwardActivate<<<outBlocks, TPB>>>(weights, outUnits, funcSignal);
}

//===========================================================================//

__global__
void feedbackDerivate(const float* signal, float* funcSignal, uint outUnits,
		float* gradient)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < outUnits)
		gradient[i] = d_derivate(funcSignal[i]) * signal[i];
}

//===========================================================================//

__global__
void feedbackSum(const float* input, float* gradient, float learning,
		uint inUnits, uint connUnits, float* weights, float* errorSignal)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = i % inUnits;
	int k = i / inUnits;

	if (i < connUnits)
	{
		weights[i] += learning * gradient[k] * input[j];
		errorSignal[j] += gradient[k] * weights[i];
		//atomicAdd(&errorSignal[j], gradient[k] * weights[i]);
	}
}

//===========================================================================//

void DeviceLayer::feedback(const float* signal, float learning)
{
	// Inicializa o sinal funcional
	hipMemset(errorSignal, 0, (inUnits - 1) * sizeof(float));

	// Calcula o gradiente
	feedbackDerivate<<<outBlocks, TPB>>>(signal, funcSignal, outUnits,
			gradient);

	// Realiza a atualização dos pesos e cálculo do sinal de erro
	feedbackSum<<<connBlocks, TPB>>>(input, gradient, learning, inUnits,
			connUnits, weights, errorSignal);
}

//===========================================================================//

uint DeviceLayer::getInUnits()
{
	return inUnits;
}

//===========================================================================//

uint DeviceLayer::getOutUnits()
{
	return outUnits;
}

//===========================================================================//

float* DeviceLayer::getFuncSignal()
{
	return funcSignal;
}

//===========================================================================//

float* DeviceLayer::getErrorSignal()
{
	return errorSignal;
}

//===========================================================================//

__device__
float d_random(hiprandState* state)
{
	float r = hiprand(state) / (float) CUDA_RAND_MAX;
	return 2 * r - 1;
}

//===========================================================================//

__device__
float d_activate(float x)
{
	return tanh(x);
}

//===========================================================================//

__device__
float d_derivate(float y)
{
	return (1 - y) * (1 + y);
}

//===========================================================================//

}

