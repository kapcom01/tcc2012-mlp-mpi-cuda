#include "hip/hip_runtime.h"
#include "mlp/cuda/DeviceExampleSet.h"

namespace ParallelMLP
{

__device__
void adjust(float &x, const Range &from, const Range &to);

//===========================================================================//

DeviceExampleSet::DeviceExampleSet(const Relation& relation)
{
	HostExampleSet set(relation);

	// Recupera os tamanhos
	size = set.getSize();
	inVars = set.getInVars();
	outVars = set.getOutVars();
	step = inVars + outVars;
	stepBlocks = (size * step) / TPB + 1;
	outBlocks = (size * outVars) / TPB + 1;

	// Aloca espaço no dispositivo
	hipMalloc(&input, size * step * sizeof(float));
	hipMalloc(&output, size * outVars * sizeof(float));
	hipMalloc(&stat, step * sizeof(Stat));

	copyToDevice(set);
}

//===========================================================================//

DeviceExampleSet::~DeviceExampleSet()
{
	hipFree(input);
	hipFree(output);
	hipFree(stat);
}

//===========================================================================//

void DeviceExampleSet::copyToDevice(const HostExampleSet &set)
{
	// Copia os dados para o dispositivo
	hipMemcpy(input, set.getInput(), size * step * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(stat, set.getStat(), step * sizeof(Stat),
			hipMemcpyHostToDevice);
}

//===========================================================================//

__global__
void normalizeVec(float* vec, Stat* stat, uint size, uint step, uint offset)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = i % step + offset;

	if (i < size * step)
		adjust(vec[i], stat[j].from, stat[j].to);
}

//===========================================================================//

void DeviceExampleSet::normalize()
{
	if (isNormalized)
		return;

	// Normaliza as entradas
	normalizeVec<<<stepBlocks, TPB>>>(input, stat, size, step, 0);

	isNormalized = true;
}

//===========================================================================//

__global__
void unnormalizeVec(float* vec, Stat* stat, uint size, uint step, uint offset)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = i % step + offset;

	if (i < size * step)
		adjust(vec[i], stat[j].to, stat[j].from);
}

//===========================================================================//

void DeviceExampleSet::unnormalize()
{
	if (!isNormalized)
		return;

	// Desnormaliza as entradas
	unnormalizeVec<<<stepBlocks, TPB>>>(input, stat, size, step, 0);

	// Desnormaliza as saídas
	unnormalizeVec<<<outBlocks, TPB>>>(output, stat, size, outVars, inVars);

	isNormalized = false;
}

//===========================================================================//

__device__
void adjust(float &x, const Range &from, const Range &to)
{
	x = (to.upper - to.lower) / (from.upper - from.lower)
			* (x - from.lower) + to.lower;
}

//===========================================================================//

uint DeviceExampleSet::getInVars() const
{
	return inVars;
}

//===========================================================================//

uint DeviceExampleSet::getOutVars() const
{
	return outVars;
}

//===========================================================================//

uint DeviceExampleSet::getSize() const
{
	return size;
}

//===========================================================================//

const float* DeviceExampleSet::getInput(uint i) const
{
	return &input[i * step];
}

//===========================================================================//

const float* DeviceExampleSet::getTarget(uint i) const
{
	return &input[i * step + inVars];
}

//===========================================================================//

void DeviceExampleSet::setOutput(uint i, float* output)
{
	float* inst = &(this->output[i * outVars]);
	hipMemcpy(inst, output, outVars * sizeof(float),
			hipMemcpyDeviceToDevice);
}

//===========================================================================//

float DeviceExampleSet::getLearning() const
{
	return learning;
}

//===========================================================================//

void DeviceExampleSet::setLearning(float learning)
{
	this->learning = learning;
}

//===========================================================================//

float DeviceExampleSet::getTolerance() const
{
	return tolerance;
}

//===========================================================================//

void DeviceExampleSet::setTolerance(float tolerance)
{
	this->tolerance = tolerance;
}

//===========================================================================//

uint DeviceExampleSet::getMaxEpochs() const
{
	return maxEpochs;
}

//===========================================================================//

void DeviceExampleSet::setMaxEpochs(uint maxEpochs)
{
	this->maxEpochs = maxEpochs;
}

//===========================================================================//

float DeviceExampleSet::getError() const
{
	return error;
}

//===========================================================================//

void DeviceExampleSet::setError(float error)
{
	this->error = error;
}

//===========================================================================//

uint DeviceExampleSet::getEpochs() const
{
	return epochs;
}

//===========================================================================//

void DeviceExampleSet::setEpochs(uint epochs)
{
	this->epochs = epochs;
}

//===========================================================================//

float DeviceExampleSet::getTime() const
{
	return time;
}

//===========================================================================//

void DeviceExampleSet::setTime(float time)
{
	this->time = time;
}

//===========================================================================//

}
